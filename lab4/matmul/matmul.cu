#include <cstdio>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
                                           
//hip/hip_runtime_api.h contains the error checking macros. note that they're called
//CUDA_CALL and CUBLAS_CALL instead of the previous names


//TODO: perform the following matrix multiplications using cublas

#define M 2 // 
#define N 3
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int main(int argc, char *argv[]) {
    double A[N * M] = {1, 2, 3, 4, 5, 6};
    double B[M * N] = {1, 2, 3, 4, 5, 6};
    double res1[N * N];
    double res2[M * M];

    double gpu_results1[N * N];
    double gpu_results2[M * M];

    int i, j, k;

    //TODO: hipMalloc buffers, copy these to device, etc.

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double *d_A, *d_B, *d_res1, *d_res2;
    double alpha = 1, beta = 0;

    hipMalloc(&d_A, N*M*sizeof(double));
    hipMalloc(&d_B, M*N*sizeof(double));
    hipMalloc(&d_res1, N*N*sizeof(double));
    hipMalloc(&d_res2, M*M*sizeof(double));


    hipMemcpy(d_A, A, N*M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*M*sizeof(double), hipMemcpyHostToDevice);

    hipMemset(d_res1, 0, N*N*sizeof(double));
    hipMemset(d_res2, 0, M*M*sizeof(double));

    // A * B
    // TODO: do this on GPU too with cuBLAS, copy result back, and printf it to check

    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                N, N, M, 
                &alpha, 
                d_A, N,
                d_B, M,
                &beta,
                d_res1, N);

    hipMemcpy(&gpu_results1, d_res1, N*N*sizeof(double), hipMemcpyDeviceToHost);

    printf("A * B\n");
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            res1[IDX2C(i, j, N)] = 0;
            for (k = 0; k < M; k++) {
                res1[IDX2C(i, j, N)] += A[IDX2C(i, k, N)] * B[IDX2C(k, j, M)];
            }
            printf("[%d, %d] = %f (cpu) and %f (gpu)\n", i, j, res1[IDX2C(i, j, N)], gpu_results1[IDX2C(i, j, N)]);
        }
    }



    // A^T * B^T
    // TODO: do this on GPU too with cuBLAS, copy result back, and printf to check it

    hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
                M, M, N,
                &alpha,
                d_A, N,
                d_B, M,
                &beta,
                d_res2, M);

    hipMemcpy(&gpu_results2, d_res2, M*M*sizeof(double), hipMemcpyDeviceToHost);

    printf("\nA^T * B^T\n");
    for (i = 0; i < M; i++) {
        for (j = 0; j < M; j++) {
            res2[IDX2C(i, j, M)] = 0;
            for (k = 0; k < N; k++) {
                res2[IDX2C(i, j, M)] += A[IDX2C(k, i, N)] * B[IDX2C(j, k, M)];
            }
            printf("[%d, %d] = %f (cpu) and %f (gpu)\n", i, j, res2[IDX2C(i, j, N)], gpu_results2[IDX2C(i, j, N)]);
        }
    }



    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_res1);
    hipFree(d_res2);

    hipblasDestroy(handle);
}