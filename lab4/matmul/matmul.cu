#include <cstdio>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
                                           
//hip/hip_runtime_api.h contains the error checking macros. note that they're called
//CUDA_CALL and CUBLAS_CALL instead of the previous names


//TODO: perform the following matrix multiplications using cublas

#define M 2 // 
#define N 3
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

int main(int argc, char *argv[]) {
    double A[N * M] = {1, 2, 3, 4, 5, 6};
    double B[M * N] = {1, 2, 3, 4, 5, 6};
    double res1[N * N];
    double res2[M * M];

    //Create output arrays for gpu matricies
    double gpu_results1[N * N];
    double gpu_results2[M * M];

    int i, j, k;

    //TODO: hipMalloc buffers, copy these to device, etc.

    //Generate handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Declare pointers
    double *d_A, *d_B, *d_res1, *d_res2;
    double alpha = 1, beta = 0;

    // Allocate Space on GPU
    hipMalloc(&d_A, N*M*sizeof(double));
    hipMalloc(&d_B, M*N*sizeof(double));
    hipMalloc(&d_res1, N*N*sizeof(double));
    hipMalloc(&d_res2, M*M*sizeof(double));

    // Transfer memory to GPU
    hipMemcpy(d_A, A, N*M*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*M*sizeof(double), hipMemcpyHostToDevice);

    // prep output arrays (not entirely necessary)
    hipMemset(d_res1, 0, N*N*sizeof(double));
    hipMemset(d_res2, 0, M*M*sizeof(double));

    // A * B
    // TODO: do this on GPU too with cuBLAS, copy result back, and printf it to check


    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, // Give handle, no operation on matricies
                N, N, M,                          // rows in output matrix, cols of output matrix, common dimension of input matricies
                &alpha,                           // Scale matrix multiplication product by one
                d_A, N,                           // Pointer to first matrix, leading dimension of first matrix
                d_B, M,                           // Pointer to second matrix, leading dimension of second matrix
                &beta,                            // Do not accumulate
                d_res1, N);                       // where to write, leading dim of output matrix

    hipMemcpy(&gpu_results1, d_res1, N*N*sizeof(double), hipMemcpyDeviceToHost);

    printf("A * B\n");
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            res1[IDX2C(i, j, N)] = 0;
            for (k = 0; k < M; k++) {
                res1[IDX2C(i, j, N)] += A[IDX2C(i, k, N)] * B[IDX2C(k, j, M)];
            }
            printf("[%d, %d] = %f (cpu) and %f (gpu)\n", i, j, res1[IDX2C(i, j, N)], gpu_results1[IDX2C(i, j, N)]);
        }
    }



    // A^T * B^T
    // TODO: do this on GPU too with cuBLAS, copy result back, and printf to check it

    hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,// Give handle, no operation on matricies
                M, M, N,// rows in output matrix, cols of output matrix, common dimension of input matricies
                &alpha,// Scale matrix multiplication product by one
                d_A, N,// Pointer to first matrix, leading dimension of first matrix
                d_B, M,// Pointer to second matrix, leading dimension of second matrix
                &beta,// Do not accumulate
                d_res2, M);// where to write, leading dim of output matrix

    for (int i = 0; i < M * N; ++i)
    {
        printf("A[%d]: %f\n", i, A[i]);
        printf("B[%d]: %f\n", i, B[i]);
    }

    hipMemcpy(&gpu_results2, d_res2, M*M*sizeof(double), hipMemcpyDeviceToHost);

    printf("\nA^T * B^T\n");
    for (i = 0; i < M; i++) {
        for (j = 0; j < M; j++) {
            res2[IDX2C(i, j, M)] = 0;
            for (k = 0; k < N; k++) {
                res2[IDX2C(i, j, M)] += A[IDX2C(k, i, N)] * B[IDX2C(j, k, M)];
            }
            printf("[%d, %d] = %f (cpu) and %f (gpu)\n", i, j, res2[IDX2C(i, j, M)], gpu_results2[IDX2C(i, j, M)]);
        }
    }



    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_res1);
    hipFree(d_res2);

    hipblasDestroy(handle);
}